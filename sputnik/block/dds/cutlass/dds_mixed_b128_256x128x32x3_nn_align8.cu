#include "sputnik/block/arguments.h"
#include "sputnik/block/dds/cutlass/dds.h"
#include "sputnik/block/cutlass/block_pitch_linear.h"
#include "sputnik/block/cutlass/default_block_gemm.h"
#include "sputnik/block/cutlass/kernel.h"
#include "sputnik/block/cutlass/threadblock_swizzle.h"
#include "sputnik/block/transpose/transpose.h"

namespace sputnik {
namespace block {
namespace cutlass {

namespace {

using dds_mixed_b128_128x128x32x5_nn_align8_base =
  typename DefaultBlockGemm<
  BlockSize::k128,
  // Non-transposed A operand.
  ::cutlass::half_t,
  ::cutlass::layout::RowMajor,
  8,
  // Non-transposed B operand.
  ::cutlass::half_t,
  BlockRowMajor,
  8,
  // C operand.
  ::cutlass::half_t,
  ::cutlass::layout::RowMajor,
  float,
  ::cutlass::arch::OpClassTensorOp,
  ::cutlass::arch::Sm80,
  ::cutlass::gemm::GemmShape<128, 128, 32>,
  ::cutlass::gemm::GemmShape<64, 64, 32>,
  ::cutlass::gemm::GemmShape<16, 8, 16>,
  ::cutlass::epilogue::thread::LinearCombination<::cutlass::half_t, 8, float, float>,
  GemmVerticalThreadblockSwizzle,
  5,
  ::cutlass::arch::OpMultiplyAdd
>::GemmKernel;

// Define named type
struct dds_mixed_b128_128x128x32x5_nn_align8 :
  public dds_mixed_b128_128x128x32x5_nn_align8_base { };

}  // namespace


bool can_launch_dds_mixed_b128_128x128x32x5_nn_align8(
    const Matrix a, bool transpose_a,
    const BlockMatrix b, bool transpose_b, Matrix c) {
  using Dds = Kernel<dds_mixed_b128_128x128x32x5_nn_align8>;

  MatmulShape shape(a, transpose_a, b, transpose_b);
  Dds::Arguments args({shape.m, shape.n, shape.k},
                      {1.0f, 0.0f},
                      {nullptr, 0},
                      {nullptr, 0},
                      {nullptr, 0},
                      {nullptr, 0});

  // Verify that we can implement the given problem.
  ::cutlass::Status status = Dds::KernelFn::can_implement(args);
  bool can_implement = status == ::cutlass::Status::kSuccess;
  can_implement &= b.block_size == BlockSize::k128;
  can_implement &= !transpose_a && !transpose_b;
  can_implement &= ValidMatmul(a, transpose_a, b, transpose_b, c);
  return can_implement;
}

hipError_t launch_dds_mixed_b128_128x128x32x5_nn_align8(
    const Matrix a, bool transpose_a,
    const BlockMatrix b, bool transpose_b,
    Matrix c, hipStream_t stream) {
  SPUTNIK_CHECK(b.offsets_t);
  SPUTNIK_CHECK(b.indices_t);
  SPUTNIK_CHECK(b.block_offsets);

  // Produce the transpose meta-data.
  if (b.create_metadata) {
    hipError_t custatus = Transpose(b, stream);
    if (custatus != hipSuccess) {
      return custatus;
    }
  }

  using Dds = Kernel<dds_mixed_b128_128x128x32x5_nn_align8>;

  MatmulShape shape(a, transpose_a, b, transpose_b);
  Dds::Arguments args({shape.m, shape.n, shape.k},
                      {1.0f, 0.0f},
                      {a.data, shape.lda},
                      {b.data,
                       b.offsets_t,
                       b.indices_t,
                       b.block_offsets,
                       shape.ldb},
                      {c.data, shape.ldc},
                      {c.data, shape.ldc});

  // Verify that we can implement the given problem.
  ::cutlass::Status status = Dds::KernelFn::can_implement(args);
  if (status != ::cutlass::Status::kSuccess) {
    return hipErrorNotSupported;
  }

  Dds dds_operator;
  return dds_operator(args, stream);
}

}  // namespace cutlass
}  // namespace block
}  // namespace sputnik
