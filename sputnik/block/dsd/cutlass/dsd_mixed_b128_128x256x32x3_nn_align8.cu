#include "sputnik/block/arguments.h"
#include "sputnik/block/dsd/cutlass/dsd.h"
#include "sputnik/block/cutlass/block_pitch_linear.h"
#include "sputnik/block/cutlass/default_block_gemm.h"
#include "sputnik/block/cutlass/kernel.h"

namespace sputnik {
namespace block {
namespace cutlass {

namespace {

using dsd_mixed_b128_128x128x32x5_nn_align8_base =
  typename DefaultBlockGemm<
  BlockSize::k128,
  // Non-transposed A operand.
  ::cutlass::half_t,
  BlockRowMajor,
  8,
  // Non-transposed B operand.
  ::cutlass::half_t,
  ::cutlass::layout::RowMajor,
  8,
  // C operand.
  ::cutlass::half_t,
  ::cutlass::layout::RowMajor,
  float,
  ::cutlass::arch::OpClassTensorOp,
  ::cutlass::arch::Sm80,
  ::cutlass::gemm::GemmShape<128, 128, 32>,
  ::cutlass::gemm::GemmShape<64, 64, 32>,
  ::cutlass::gemm::GemmShape<16, 8, 16>,
  ::cutlass::epilogue::thread::LinearCombination<::cutlass::half_t, 8, float, float>,
  ::cutlass::gemm::threadblock::GemmHorizontalThreadblockSwizzle,
  5,
  ::cutlass::arch::OpMultiplyAdd
>::GemmKernel;

// Define named type
struct dsd_mixed_b128_128x128x32x5_nn_align8 :
  public dsd_mixed_b128_128x128x32x5_nn_align8_base { };

}  // namespace


bool can_launch_dsd_mixed_b128_128x128x32x5_nn_align8(
    const BlockMatrix a, bool transpose_a,
    const Matrix b, bool transpose_b, Matrix c) {
  using Dsd = Kernel<dsd_mixed_b128_128x128x32x5_nn_align8>;

  MatmulShape shape(a, transpose_a, b, transpose_b);
  Dsd::Arguments args({shape.m, shape.n, shape.k},
                      {1.0f, 0.0f},
                      {nullptr, 0},
                      {nullptr, 0},
                      {nullptr, 0},
                      {nullptr, 0});

  // Verify that we can implement the given problem.
  ::cutlass::Status status = Dsd::KernelFn::can_implement(args);
  bool can_implement = status == ::cutlass::Status::kSuccess;
  can_implement &= a.block_size == BlockSize::k128;
  can_implement &= !transpose_a && !transpose_b;
  can_implement &= ValidMatmul(a, transpose_a, b, transpose_b, c);
  return can_implement;
}

hipError_t launch_dsd_mixed_b128_128x128x32x5_nn_align8(
    const BlockMatrix a, bool transpose_a,
    const Matrix b, bool transpose_b,
    Matrix c, hipStream_t stream) {
  using Dsd = Kernel<dsd_mixed_b128_128x128x32x5_nn_align8>;

  MatmulShape shape(a, transpose_a, b, transpose_b);
  Dsd::Arguments args({shape.m, shape.n, shape.k},
                      {1.0f, 0.0f},
                      {a.data, a.offsets, a.indices, shape.lda},
                      {b.data, shape.ldb},
                      {c.data, shape.ldc},
                      {c.data, shape.ldc});

  // Verify that we can implement the given problem.
  ::cutlass::Status status = Dsd::KernelFn::can_implement(args);
  if (status != ::cutlass::Status::kSuccess) {
    return hipErrorNotSupported;
  }

  Dsd dsd_operator;
  return dsd_operator(args, stream);
}

}  // namespace cutlass
}  // namespace block
}  // namespace sputnik
