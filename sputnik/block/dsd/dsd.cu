#include <iostream>

#include "sputnik/block/dsd/dsd.h"
#include "sputnik/block/dsd/cutlass/dsd.h"

namespace sputnik {
namespace block {

hipError_t Matmul(
    const BlockMatrix a, bool transpose_a,
    const Matrix b, bool transpose_b,
    Matrix c, hipStream_t stream) {
  if (a.block_size == BlockSize::k128) {
    return cutlass::Matmul(a, transpose_a, b, transpose_b, c, stream);
  }
  return hipErrorNotSupported;
}


hipError_t MatmulEx(
    const BlockMatrix a, bool transpose_a,
    const Matrix b, bool transpose_b,
    Matrix c, hipStream_t stream) {
  BlockMatrix acp = a;
  acp.create_metadata = false;
  return Matmul(acp, transpose_a, b, transpose_b, c, stream);
}
  
}  // namespace block
}  // namespace sputnik
