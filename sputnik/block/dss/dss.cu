#include <iostream>

#include "sputnik/block/dss/dss.h"
#include "sputnik/block/dss/cutlass/dss.h"

namespace sputnik {
namespace block {

hipError_t Matmul(
    const BlockMatrix a, bool transpose_a,
    const BlockMatrix b, bool transpose_b,
    Matrix c, hipStream_t stream) {
  return cutlass::Matmul(a, transpose_a, b, transpose_b, c, stream);
}

hipError_t MatmulEx(
    const BlockMatrix a, bool transpose_a,
    const BlockMatrix b, bool transpose_b,
    Matrix c, hipStream_t stream) {
  BlockMatrix acp = a, bcp = b;
  acp.create_metadata = false;
  bcp.create_metadata = false;
  return cutlass::Matmul(acp, transpose_a, bcp, transpose_b, c, stream);
}

}  // namespace block
}  // namespace sputnik
