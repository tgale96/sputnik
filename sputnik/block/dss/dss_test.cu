// Copyright 2020 The Sputnik Authors.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
#include <iostream>

#include "sputnik/cuda_utils.h"
#include "sputnik/block/dss/dss.h"
#include "sputnik/block/bitmask/bitmask.h"
#include "sputnik/block/matrix_utils.h"

#include "absl/random/random.h"
#include "gmock/gmock.h"
#include "gtest/gtest.h"

namespace sputnik {
namespace block {

using ::testing::NanSensitiveFloatNear;
using ::testing::Pointwise;

template <
    int kDimM_,
    int kDimK_,
    int kDimN_,
    int kNonZerosA_,
    int kNonZerosB_,
    int kBlockDim_,
    bool kTransposeA_ = false,
    bool kTransposeB_ = false,
    bool kUnorderedIndices_ = false>
struct Problem {
  static_assert(kNonZerosA_ <= kDimM_ * kDimK_,
                "Number of non-zero must fit in the lhs matrix.");
  static_assert(kNonZerosB_ <= kDimK_ * kDimN_,
                "Number of non-zero must fit in the rhs matrix.");

  static constexpr int kDimM = kDimM_;
  static constexpr int kDimK = kDimK_;
  static constexpr int kDimN = kDimN_;
  static constexpr int kNonZerosA = kNonZerosA_;
  static constexpr int kNonZerosB = kNonZerosB_;
  static constexpr int kBlockDim = kBlockDim_;
  static constexpr int kTransposeA = kTransposeA_;
  static constexpr int kTransposeB = kTransposeB_;
  static constexpr bool kUnorderedIndices = kUnorderedIndices_;
};

template <typename Problem>
class DssTest : public ::testing::Test {
 public:
  const int kDimM = Problem::kDimM;
  const int kDimK = Problem::kDimK;
  const int kDimN = Problem::kDimN;
  const int kNonZerosA = Problem::kNonZerosA;
  const int kNonZerosB = Problem::kNonZerosB;
  const int kBlockDim = Problem::kBlockDim;
  const int kTransposeA = Problem::kTransposeA;
  const int kTransposeB = Problem::kTransposeB;
  const bool kUnorderedIndices = Problem::kUnorderedIndices;

  // Random number generator for creating matrices.
  absl::BitGen generator_;
};

// TODO(tgale): Make our test harness more efficient so that we
// can test larger problems (up to 32k).
typedef ::testing::Types<
    // Block 128 problems NN.
    Problem<128, 128, 128, 128*128, 128*128, 128>,  // Minimum problem size.
    Problem<128, 256, 128, 128*256, 128*256, 128>,  // Two inner loops.
    Problem<256, 128, 128, 128*256, 128*128, 128>,  // Two rows of blocks.
    Problem<128, 128, 256, 128*128, 128*256, 128>,  // Two columns of blocks.
    Problem<128, 256, 128, 128*128, 128*256, 128>,  // 50% sparse lhs.
    Problem<128, 256, 128, 128*256, 128*128, 128>,  // 50% sparse rhs.
    Problem<128, 256, 128, 128*128, 128*128, 128>,  // 50% sparse both.
    Problem<256, 128, 128, 128*128, 128*128, 128>,  // 50% lhs, multi-row.
    Problem<128, 128, 256, 128*128, 128*128, 128>,  // 50% rhs, multi-col.
    Problem<256, 128, 128, 128*128, 128*128, 128>,  // 50% both, multi-both.
    Problem<256, 256, 256, 128*256, 256*128, 128>,  // 50% both, two loops.
    // Block 128 problems NT.
    Problem<128, 128, 128, 128*128, 128*128, 128, false, true>,
    Problem<128, 256, 128, 128*256, 128*256, 128, false, true>,
    Problem<256, 128, 128, 128*256, 128*128, 128, false, true>,
    Problem<128, 128, 256, 128*128, 128*256, 128, false, true>,
    Problem<128, 256, 128, 128*128, 128*256, 128, false, true>,
    Problem<128, 256, 128, 128*256, 128*128, 128, false, true>,
    Problem<128, 256, 128, 128*128, 128*128, 128, false, true>,
    Problem<256, 128, 128, 128*128, 128*128, 128, false, true>,
    Problem<128, 128, 256, 128*128, 128*128, 128, false, true>,
    Problem<256, 128, 128, 128*128, 128*128, 128, false, true>,
    Problem<256, 256, 256, 128*256, 256*128, 128, false, true>,
    // Block 128 problems TN.
    Problem<128, 128, 128, 128*128, 128*128, 128, true>,
    Problem<128, 256, 128, 128*256, 128*256, 128, true>,
    Problem<256, 128, 128, 128*256, 128*128, 128, true>,
    Problem<128, 128, 256, 128*128, 128*256, 128, true>,
    Problem<128, 256, 128, 128*128, 128*256, 128, true>,
    Problem<128, 256, 128, 128*256, 128*128, 128, true>,
    Problem<128, 256, 128, 128*128, 128*128, 128, true>,
    Problem<256, 128, 128, 128*128, 128*128, 128, true>,
    Problem<128, 128, 256, 128*128, 128*128, 128, true>,
    Problem<256, 128, 128, 128*128, 128*128, 128, true>,
    Problem<256, 256, 256, 128*256, 256*128, 128, true>,
    // Block 128 problems TT.
    Problem<128, 128, 128, 128*128, 128*128, 128, true, true>,
    Problem<128, 256, 128, 128*256, 128*256, 128, true, true>,
    Problem<256, 128, 128, 128*256, 128*128, 128, true, true>,
    Problem<128, 128, 256, 128*128, 128*256, 128, true, true>,
    Problem<128, 256, 128, 128*128, 128*256, 128, true, true>,
    Problem<128, 256, 128, 128*256, 128*128, 128, true, true>,
    Problem<128, 256, 128, 128*128, 128*128, 128, true, true>,
    Problem<256, 128, 128, 128*128, 128*128, 128, true, true>,
    Problem<128, 128, 256, 128*128, 128*128, 128, true, true>,
    Problem<256, 128, 128, 128*128, 128*128, 128, true, true>,
    Problem<256, 256, 256, 128*256, 256*128, 128, true, true>,
    // Larger problems NN.
    Problem<512, 512, 512, 512*512, 512*512, 128>,
    Problem<512, 512, 512, 256*512, 256*512, 128>,
    Problem<512, 512, 512, 128*512, 128*512, 128>,
    Problem<1024, 1024, 1024, 1024*1024, 1024*1024, 128>,
    Problem<1024, 1024, 1024, 512*1024, 512*1024, 128>,
    Problem<1024, 1024, 1024, 256*1024, 256*1024, 128>,
    // Larger problems NT.
    Problem<512, 512, 512, 512*512, 512*512, 128, false, true>,
    Problem<512, 512, 512, 256*512, 256*512, 128, false, true>,
    Problem<512, 512, 512, 128*512, 128*512, 128, false, true>,
    Problem<1024, 1024, 1024, 1024*1024, 1024*1024, 128, false, true>,
    Problem<1024, 1024, 1024, 512*1024, 512*1024, 128, false, true>,
    Problem<1024, 1024, 1024, 256*1024, 256*1024, 128, false, true>,
    // Larger problems TN.
    Problem<512, 512, 512, 512*512, 512*512, 128, true>,
    Problem<512, 512, 512, 256*512, 256*512, 128, true>,
    Problem<512, 512, 512, 128*512, 128*512, 128, true>,
    Problem<1024, 1024, 1024, 1024*1024, 1024*1024, 128, true>,
    Problem<1024, 1024, 1024, 512*1024, 512*1024, 128, true>,
    Problem<1024, 1024, 1024, 256*1024, 256*1024, 128, true>,
    // Larger problems TT.
    Problem<512, 512, 512, 512*512, 512*512, 128, true, true>,
    Problem<512, 512, 512, 256*512, 256*512, 128, true, true>,
    Problem<512, 512, 512, 128*512, 128*512, 128, true, true>,
    Problem<1024, 1024, 1024, 1024*1024, 1024*1024, 128, true, true>,
    Problem<1024, 1024, 1024, 512*1024, 512*1024, 128, true, true>,
    Problem<1024, 1024, 1024, 256*1024, 256*1024, 128, true, true>,
    // Unordered problems NN.
    Problem<512, 512, 512, 512*512, 512*512, 128, false, false, true>,
    Problem<512, 512, 512, 256*512, 256*512, 128, false, false, true>,
    Problem<512, 512, 512, 128*512, 128*512, 128, false, false, true>,
    Problem<1024, 1024, 1024, 1024*1024, 1024*1024, 128, false, false, true>,
    Problem<1024, 1024, 1024, 512*1024, 512*1024, 128, false, false, true>,
    Problem<1024, 1024, 1024, 256*1024, 256*1024, 128, false, false, true>,
    // Unordered problems TN.
    Problem<512, 512, 512, 512*512, 512*512, 128, true, false, true>,
    Problem<512, 512, 512, 256*512, 256*512, 128, true, false, true>,
    Problem<512, 512, 512, 128*512, 128*512, 128, true, false, true>,
    Problem<1024, 1024, 1024, 1024*1024, 1024*1024, 128, true, false, true>,
    Problem<1024, 1024, 1024, 512*1024, 512*1024, 128, true, false, true>,
    Problem<1024, 1024, 1024, 256*1024, 256*1024, 128, true, false, true>,
    // Unordered problems NT.
    Problem<512, 512, 512, 512*512, 512*512, 128, false, true, true>,
    Problem<512, 512, 512, 256*512, 256*512, 128, false, true, true>,
    Problem<512, 512, 512, 128*512, 128*512, 128, false, true, true>,
    Problem<1024, 1024, 1024, 1024*1024, 1024*1024, 128, false, true, true>,
    Problem<1024, 1024, 1024, 512*1024, 512*1024, 128, false, true, true>,
    Problem<1024, 1024, 1024, 256*1024, 256*1024, 128, false, true, true>,
    // Unordered problems TT.
    Problem<512, 512, 512, 512*512, 512*512, 128, true, true, true>,
    Problem<512, 512, 512, 256*512, 256*512, 128, true, true, true>,
    Problem<512, 512, 512, 128*512, 128*512, 128, true, true, true>,
    Problem<1024, 1024, 1024, 1024*1024, 1024*1024, 128, true, true, true>,
    Problem<1024, 1024, 1024, 512*1024, 512*1024, 128, true, true, true>,
    Problem<1024, 1024, 1024, 256*1024, 256*1024, 128, true, true, true>,
    > TestProblems;

TYPED_TEST_SUITE(DssTest, TestProblems);

TYPED_TEST(DssTest, Dss) {
  // Create the lhs matrix on cpu & gpu.
  int oda = this->kTransposeA ? this->kDimK : this->kDimM;
  int lda = this->kTransposeA ? this->kDimM : this->kDimK;
  BlockSparseMatrix lhs_(
      oda, lda, this->kNonZerosA, this->kBlockDim,
      RANDOM_UNIFORM, &this->generator_,
      /*pad_rows_to=*/1,
      /*unordered_indices=*/this->kUnorderedIndices);
  sputnik::Matrix lhs = ToMatrix(lhs_);
  CudaBlockSparseMatrix<half> lhs_gpu(lhs_);

  // Create the rhs matrix on cpu & gpu
  int odb = this->kTransposeB ? this->kDimN : this->kDimK;
  int ldb = this->kTransposeB ? this->kDimK : this->kDimN;
  BlockSparseMatrix rhs_(
      odb, ldb, this->kNonZerosB, this->kBlockDim,
      RANDOM_UNIFORM, &this->generator_,
      /*pad_rows_to=*/1,
      /*unordered_indices=*/this->kUnorderedIndices);
  sputnik::Matrix rhs = ToMatrix(rhs_);
  CudaBlockSparseMatrix<half> rhs_gpu(rhs_);

  // Create the output matrix on gpu & gpu.
  CudaMatrix<half> out_gpu(this->kDimM, this->kDimN, &this->generator_);

  // Run the gpu kernel.
  BlockMatrix lhs_args = Arg(lhs_gpu);
  BlockMatrix rhs_args = Arg(rhs_gpu);
  if (this->kTransposeA) AllocateTransposeBuffers(lhs_args);
  if (!this->kTransposeB) AllocateTransposeBuffers(rhs_args);
  AllocateBitmaskBuffers(lhs_args);
  AllocateBitmaskBuffers(rhs_args);
  CUDA_CALL(Matmul(lhs_args, this->kTransposeA,
                   rhs_args, this->kTransposeB,
                   Arg(out_gpu), /*stream=*/0));
  CUDA_CALL(hipStreamSynchronize(nullptr));
  FreeBitmaskBuffers(lhs_args);
  FreeBitmaskBuffers(rhs_args);
  if (this->kTransposeA) FreeTransposeBuffers(lhs_args);
  if (!this->kTransposeB) FreeTransposeBuffers(rhs_args);

  // Verify the results.
  sputnik::Matrix expected =
      (this->kTransposeA ? lhs.T() : lhs) *
      (this->kTransposeB ? rhs.T() : rhs);
  sputnik::Matrix results(out_gpu);
  auto comparator = Pointwise(NanSensitiveFloatNear(5e-02), ToVector(expected));
  ASSERT_THAT(ToVector(results), comparator);
}

}  // namespace block
}  // namespace sputnik
