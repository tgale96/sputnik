#include "sputnik/timer.h"

namespace sputnik {

Timer::Timer() {
  for (auto &event : events) {
    CUDA_CALL(hipEventCreate(&event));
  }
}

Timer::~Timer() {
  for (auto &event : events) {
    CUDA_CALL(hipEventDestroy(event));
  }
}

void Timer::start(hipStream_t stream) {
  CUDA_CALL(hipEventRecord(events[0], stream));
}

/// Records a stop event in the stream and synchronizes on the stream
void Timer::stop(hipStream_t stream) {
  CUDA_CALL(hipEventRecord(events[1], stream));
  CUDA_CALL(hipStreamSynchronize(stream));
}

/// Returns the duration in miliseconds
double Timer::duration(int iterations) const {
  float avg_ms;
  CUDA_CALL(hipEventElapsedTime(&avg_ms, events[0], events[1]));
  return (double)avg_ms / iterations;
}

}  // namespace sputnik
